#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

void check(hipError_t x) {
  //  fprintf(stderr, "%s\n", cudaGetErrorString(x));
}

void showMatrix2(char* v1, int width, int height) {
  //  printf("---------------------\n");
   
    for (int i = 0; i < width; i++) {
	if (i != 0){
		printf(" %d\n", i);}
        for (int j = 0; j < height; j++) {
            printf("%c", v1[i * width + j]);	
        }
	if (i == 11){
	    printf(" %d\n ", i+1);}
       // printf("\n");
    }
}

__global__ void kernel(char* tab,int width, int height, int pitch) {

    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < width && col < height) {
        *( ((int *)(((char *)tab) + (row * pitch))) + col) = '&';
    }
}

int main()
{
    int width = 12;
    int height = 156;

    char* d_tab;
    char* h_tab;

    int realSize = width * height* sizeof(int);

    size_t pitch;
    check( hipMallocPitch(&d_tab, &pitch, width * sizeof(int), height) );
    h_tab = (char*)malloc(realSize);
    check( hipMemset(d_tab, 0, realSize) );

    dim3 grid(39, 12);
    dim3 block(39, 12);
    kernel <<<grid, block>>>(d_tab, width, height, pitch);

    check( hipMemcpy2D(h_tab, width*sizeof(int), d_tab, pitch, width*sizeof(int), height, hipMemcpyDeviceToHost) );

    showMatrix2(h_tab, width, height);
   // printf("\nPitch size: %ld\n", pitch);
    return 0;
}
